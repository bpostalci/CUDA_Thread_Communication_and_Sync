// Berat Postalcioglu
/* OUTPUT

	minimum element of the array (minCPU): -9649.35
	minimum element of the array (minGPU): -9649.35

*/
#include "hip/hip_runtime.h"

#include <cmath>
#include <cstdio>
#include <ctime>

const int ArrSize = 60000;
const int ThreadsPerBlock = 512;
const int BlocksPerGrid = 32;

// generates a random array
void generateArray(double *data, int count) {
	//generate a random data set
	for (int i = 0; i < count; i++) {
		data[i] = rand() / (rand() + 1.1) * (rand() % 2 ? 1 : -1);
	}

}

double minCPU(double *data, int count)
{
	int minIndex = 0;
	for (int i = 0; i < count; i++)
	{
		if (std::isgreater(data[minIndex], data[i]))
		{
			minIndex = i;
		}
	}
	return data[minIndex];
}

__global__ void minGPU(double *data, int count, double *res)
{
	__shared__ double cache[ThreadsPerBlock];

	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int cacheIndex = threadIdx.x;
	double temp = 0;

	while (tid < count)
	{
		temp += data[tid];
		//cache[cacheIndex] = data[tid];
		tid += blockDim.x * gridDim.x;
	}
	cache[cacheIndex] = temp;

	__syncthreads();

	int i = blockDim.x / 2;
	while (i != 0) {
		if (cacheIndex < i)
		{
			if (cache[cacheIndex] > cache[cacheIndex + i])
			{
				cache[cacheIndex] = cache[cacheIndex + i];
			}
		}			
		__syncthreads();
		i /= 2;
	}
	if (cacheIndex == 0)
		res[blockIdx.x] = cache[0];
	
}

int main()
{
	srand(time(NULL));

	// cpu
	double data[ArrSize];
	generateArray(data, ArrSize);
	double minElementCpu = minCPU(data, ArrSize);
	printf("minimum element of the array (minCPU): %.2f\n", minElementCpu);

	// gpu
	double *gpuData, *gpuRes;
	hipMalloc((void**)&gpuData, ArrSize * sizeof(double));
	hipMalloc((void**)&gpuRes, BlocksPerGrid * sizeof(double));
	hipMemcpy((void*)gpuData, (const void*) data, ArrSize * sizeof(double), hipMemcpyHostToDevice);
	minGPU <<<BlocksPerGrid, ThreadsPerBlock>>> (gpuData, ArrSize, gpuRes);

	double blockResults[BlocksPerGrid];
	hipMemcpy((void*)blockResults, (const void *)gpuRes, BlocksPerGrid * sizeof(double), hipMemcpyDeviceToHost);

	double minElementGpu = minCPU(blockResults, BlocksPerGrid);
	printf("minimum element of the array (minGPU): %.2f\n", minElementGpu);
	
	return 0;
}